#include <stdio.h>
#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include <sys/time.h>

#define N_THREADS 256
// To check for nasty errors
#define CHECK(call) {                                                   \
        const hipError_t error = call;                                 \
        if(error != hipSuccess){                                       \
            printf("Error: %s:%d, ", __FILE__, __LINE__);               \
            printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
            exit(-10*error);                                            \
        }                                                               \
    }

// GPU pointers
unsigned char *si_cuda;
unsigned char *ri_cuda;
unsigned long long *tmp_cuda;
unsigned long long *d_cuda;

/**
 * Measure time
 */
double cpuSeconds(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
}

/**
 * GPU kernel
 */
__global__ void GPUDistance(const unsigned char *si, const unsigned char *ri, unsigned long long *d, unsigned long long *tmp_cuda, const int size){
    int index = (blockDim.x * blockIdx.x) + threadIdx.x; // Global index
    int tid = threadIdx.x; // local index
    unsigned long long tmp;
    unsigned long long *ldata = tmp_cuda + (blockDim.x * blockIdx.x); // local pointer to global memory

    // check boundary
    if(index >= size)
        return;

    // Diff images
    tmp = si[index] - ri[index];
    tmp_cuda[index] = tmp * tmp;

    // reduce: sum the diff vector
    for(int stride = blockDim.x >> 1; stride > 0; stride >>= 1){
        __syncthreads();
        if(tid < stride)
            ldata[tid] += ldata[tid + stride];
    }
    __syncthreads(); // needed?

    // copy the block result.
    if(tid == 0){
        d[blockIdx.x] = ldata[0];
    }
}

/**
 * Copy source image to GPU
 */
void CopySourceImage(const unsigned char *image, const int size){
    hipMemcpy(si_cuda, image, size, hipMemcpyHostToDevice);
}

/**
 * Copy test image to GPU
 */
void CopyRenderImage(const unsigned char *image, const int size){
    hipMemcpy(ri_cuda, image, size, hipMemcpyHostToDevice);
}

/**
 * Alloc GPU memory, and set to zero.
 * The number of bytes to alloc is a multiple of N_THREADS.
 */
void MallocGPUMemory(const int size){
    int n_blocks = (size + N_THREADS - 1)/N_THREADS;
    int size2 = n_blocks * N_THREADS;
    CHECK(hipMalloc((void **)&si_cuda, size2));
    CHECK(hipMalloc((void **)&ri_cuda, size2));
    CHECK(hipMalloc((void **)&tmp_cuda, size2 * sizeof(unsigned long long)));
    CHECK(hipMalloc((void **)&d_cuda, n_blocks * sizeof(unsigned long long)));

    // clear memory
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemset((void *)si_cuda, 0, size2));
    CHECK(hipMemset((void *)ri_cuda, 0, size2));
    CHECK(hipMemset((void *)tmp_cuda, 0, size2 * sizeof(unsigned long long)));
    CHECK(hipMemset((void *)d_cuda, 0, n_blocks * sizeof(unsigned long long)));
    printf("CUDA memory created and set to 0\n\n");
}

/**
 * Free the GPU memory
 */
void FreeGPUMemory(void){
     hipFree(si_cuda);
     hipFree(ri_cuda);
     hipFree(tmp_cuda);
     hipFree(d_cuda);
}

/**
 * Launch the GPU kernel, and performs the final reduction in the result vector.
 */
unsigned long long DistanceGPU(unsigned char *ri, const int size){
    int threadsPerBlock = N_THREADS;
    int blocksPerGrid = (size + threadsPerBlock - 1)/threadsPerBlock;
    unsigned long long distance[blocksPerGrid];
    unsigned long long tmp = 0L;

    int n_blocks = (size + N_THREADS - 1)/N_THREADS;
    int size2 = n_blocks * N_THREADS;

    CopyRenderImage(ri, size);

    GPUDistance<<<blocksPerGrid, threadsPerBlock>>>(si_cuda, ri_cuda, d_cuda, tmp_cuda, size2);
    hipDeviceSynchronize();

    hipMemcpy(distance, d_cuda, blocksPerGrid * sizeof(unsigned long long), hipMemcpyDeviceToHost);
    for(int i = 0; i < blocksPerGrid; i++)
        tmp += distance[i];
    return tmp;
}
